
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void setTrue(bool* status) { *status = true; }

void check(bool const* status) {
  std::cout << "sizeof(bool) is " << sizeof(bool) << '\n';
  std::cout << "value of status is " << static_cast<int>(*reinterpret_cast<const char*>(status)) << '\n';
  std::cout << "status is " << std::noboolalpha << *status << " (" << std::boolalpha << *status << ")\n";
  std::cout << "status == true:  " << (*status == true) << '\n';
  std::cout << "status != true:  " << (*status != true) << '\n';
  std::cout << "status == false: " << (*status == false) << '\n';
  std::cout << "status != false: " << (*status != false) << '\n';
  std::cout << '\n';
}

int main() {
  bool* status;
  hipHostMalloc(&status, sizeof(bool), hipHostMallocDefault);
  *status = false;
  check(status);

  setTrue<<<1, 1>>>(status);
  hipDeviceSynchronize();
  check(status);

  *status = true;
  check(status);
}
